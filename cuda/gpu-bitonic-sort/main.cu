#define DEBUG

#include "hip/hip_runtime.h"

#include <stdio.h>

#define ARR_SIZE 100000
#define BLOCK_SIZE 1024

__global__ void sort (long long* data)
{
    long long arrSizeCopy = ARR_SIZE;
    long long operations = 0;
    while (arrSizeCopy > 0) {
        arrSizeCopy = arrSizeCopy >> 1;
        ++operations;
    }
    long long fakeArrSize = 1 << operations;

    for (int i = 0; i < operations; ++i) {
        long long rectSize = 1 << (i + 1);
        long long halfRectSize = rectSize >> 1;

        while (rectSize > 1) {

            for (int iElement = threadIdx.x; iElement < fakeArrSize; iElement += BLOCK_SIZE) {
                // -1 - смотрит в начало, 1 - в конец
                int direction = -1;
                if ((iElement / rectSize) % 2 == 0) {
                    direction = 1;
                }

                // 0 - половина большая к началу, 1 - к концу
                int half = 1;
                if (iElement % rectSize < rectSize / 2) {
                    half = 0;
                }

                if ((direction == 1) && (half == 0)) {
                    if ((iElement < ARR_SIZE) && (iElement + halfRectSize < ARR_SIZE)) {
                        if (data[iElement] > data[iElement + halfRectSize]) {
                            long long tmp = data[iElement + halfRectSize];
                            data[iElement + halfRectSize] = data[iElement];
                            data[iElement] = tmp;
                        }
                    }
                }

                if ((direction == -1) && (half == 1)) {
                    if ((iElement < ARR_SIZE) && (iElement - halfRectSize < ARR_SIZE)) {
                        if (data[iElement] > data[iElement - halfRectSize]) {
                            long long tmp = data[iElement - halfRectSize];
                            data[iElement - halfRectSize] = data[iElement];
                            data[iElement] = tmp;
                        }
                    }
                }

            }

            __syncthreads();

            rectSize = rectSize >> 1;
            halfRectSize = rectSize >> 1;
        }
    }




    long long operationOnThread = ARR_SIZE / BLOCK_SIZE / 2 + 1;

    for (long long i = 0; i < ARR_SIZE / 2 + 1; ++i) {

        for (long long iOperation = 0; iOperation < operationOnThread; ++iOperation) {
            long long realIndex = (BLOCK_SIZE * iOperation + threadIdx.x) * 2;
            long long nextIndex = realIndex + 1;
            if (nextIndex < ARR_SIZE) {
                if (data[realIndex] > data[nextIndex]) {
                    long long tmp = data[realIndex];
                    data[realIndex] = data[nextIndex];
                    data[nextIndex] = tmp;
                }
            }
        }

        __syncthreads();

        for (long long iOperation = 0; iOperation < operationOnThread; ++iOperation) {
            long long realIndex = (BLOCK_SIZE * iOperation + threadIdx.x) * 2 + 1;
            long long nextIndex = realIndex + 1;
            if (nextIndex < ARR_SIZE) {
                if (data[realIndex] > data[nextIndex]) {
                    long long tmp = data[realIndex];
                    data[realIndex] = data[nextIndex];
                    data[nextIndex] = tmp;
                }
            }
        }

        __syncthreads();

    }
}

int main()
{
    long long arr[1000000];

    FILE* f = fopen("../input.txt", "rt");
    long long readIndex = 0;
    long long temp = 0;
    while (fscanf(f, "%lld", &temp) == 1) {
        arr[readIndex] = temp;
        ++readIndex;
    }

    #if defined(DEBUG)
        printf("\nИзначальный массив:\n");
        printf("Размер массива: %lld\n", ARR_SIZE);
        long long sizeForPrintf = 100;
        if (sizeForPrintf > ARR_SIZE) {
            sizeForPrintf = ARR_SIZE;
        }
        printf("Массив/Первые 100 его элементов:\n");
        for (long long i = 0; i < sizeForPrintf; ++i) {
            printf("%lld ", arr[i]);
        }
        printf("\n");
    #endif

    long long* array;
    hipMalloc(&array, ARR_SIZE * sizeof(long long));

    hipMemcpy(array, arr, ARR_SIZE * sizeof(long long), hipMemcpyHostToDevice);


    hipEvent_t start, stop;
    float milliseconds;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);


    sort<<< 1, BLOCK_SIZE >>>(array);
    hipMemcpy(arr, array, ARR_SIZE * sizeof(long long), hipMemcpyDeviceToHost);

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    double seconds = (double) milliseconds / 1000;
    printf("Заняло %f секунд\n", seconds);
    hipEventDestroy(start);
    hipEventDestroy(stop);


    #if defined(DEBUG)
        printf("\nОтсортированный массив:\n");
        printf("Массив/Первые 100 его элементов:\n");
        for (long long i = 0; i < sizeForPrintf; ++i) {
            printf("%lld ", arr[i]);
        }
        printf("\n");
        printf("Потребовалось операций: Потом будет\n");
    #endif

    return 0;
}
