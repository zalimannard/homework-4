#define FULL

#include "hip/hip_runtime.h"

#include <stdio.h>

#define ARR_SIZE (long int) 1024
#define BLOCK_SIZE 1024

__global__ void sort (long int* data, unsigned long long* operations)
{
    long int arrSizeCopy = ARR_SIZE;
    int iterations = 0;
    while (arrSizeCopy > 0) {
        arrSizeCopy = arrSizeCopy >> 1;
        ++iterations;
    }
    long int fakeArrSize = 1 << iterations;
    int direction = 0;
    int half = 0;
    long int tmp = 0;

    for (int i = 0; i < iterations; ++i) {
        #if defined(FULL)
            atomicAdd(operations, (unsigned long long) 2);
        #endif
        long int rectSize = 1 << (i + 1);
        long int halfRectSize = rectSize >> 1;

        long int stableRectSize = rectSize;
        while (rectSize > 1) {
            #if defined(FULL)
                atomicAdd(operations, (unsigned long long) 1);
            #endif
            for (long int iElement = threadIdx.x; iElement < fakeArrSize; iElement += BLOCK_SIZE) {
                #if defined(FULL)
                    atomicAdd(operations, (unsigned long long) 2);
                #endif
                // -1 - смотрит в начало, 1 - в конец
                direction = -1;
                if ((iElement / stableRectSize) % 2 == 0) {
                    direction = 1;
                    #if defined(FULL)
                        atomicAdd(operations, (unsigned long long) 1);
                    #endif
                }
                #if defined(FULL)
                    atomicAdd(operations, (unsigned long long) 4);
                #endif

                // 0 - половина большая к началу, 1 - к концу
                half = 1;
                if (iElement % rectSize < rectSize / 2) {
                    half = 0;
                    #if defined(FULL)
                        atomicAdd(operations, (unsigned long long) 1);
                    #endif
                }
                #if defined(FULL)
                    atomicAdd(operations, (unsigned long long) 4);
                #endif

                if ((direction == 1) && (half == 0)) {
                    if ((iElement < ARR_SIZE) && (iElement + halfRectSize < ARR_SIZE)) {
                        if (data[iElement] > data[iElement + halfRectSize]) {
                            tmp = data[iElement + halfRectSize];
                            data[iElement + halfRectSize] = data[iElement];
                            data[iElement] = tmp;
                            #if defined(FULL)
                                atomicAdd(operations, (unsigned long long) 3);
                            #endif
                        }
                        #if defined(FULL)
                            atomicAdd(operations, (unsigned long long) 3);
                        #endif
                    }
                    #if defined(FULL)
                        atomicAdd(operations, (unsigned long long) 3);
                    #endif
                }
                #if defined(FULL)
                    atomicAdd(operations, (unsigned long long) 2);
                #endif

                if ((direction == -1) && (half == 1)) {
                    if ((iElement < ARR_SIZE) && (iElement - halfRectSize < ARR_SIZE)) {
                        if (data[iElement] > data[iElement - halfRectSize]) {
                            tmp = data[iElement - halfRectSize];
                            data[iElement - halfRectSize] = data[iElement];
                            data[iElement] = tmp;
                            #if defined(FULL)
                                atomicAdd(operations, (unsigned long long) 3);
                            #endif
                        }
                        #if defined(FULL)
                            atomicAdd(operations, (unsigned long long) 3);
                        #endif
                    }
                    #if defined(FULL)
                        atomicAdd(operations, (unsigned long long) 3);
                    #endif
                }
                #if defined(FULL)
                    atomicAdd(operations, (unsigned long long) 2);
                #endif
            }

            __syncthreads();

            rectSize = rectSize >> 1;
            halfRectSize = rectSize >> 1;
        }
    }
}

int main()
{
    printf("Параллельная сортировка пузырьком на GPU\n");
    printf("Размер массива: %ld\n", ARR_SIZE);

    #if defined(FULL)
        printf("Запуск в медленном режиме\n\n");
    #else
        printf("Запуск в быстром режиме\n\n");
    #endif

    long int arr[ARR_SIZE];

    FILE* f = fopen("../input.txt", "rt");
    long int readIndex = 0;
    long int temp = 0;
    while (fscanf(f, "%ld", &temp) == 1) {
        arr[readIndex] = temp;
        ++readIndex;
    }

    #if defined(FULL)
        printf("\nИзначальный массив:\n");
        long int sizeForPrintf = 100;
        if (sizeForPrintf > ARR_SIZE) {
            sizeForPrintf = ARR_SIZE;
        }
        printf("Массив/Первые 100 его элементов:\n");
        for (long int i = 0; i < sizeForPrintf; ++i) {
            printf("%ld ", arr[i]);
        }
        printf("\n");
    #endif

    long int* array;
    hipMalloc(&array, ARR_SIZE * sizeof(long int));
    hipMemcpy(array, arr, ARR_SIZE * sizeof(long int), hipMemcpyHostToDevice);

    unsigned long long operations = 0;
    unsigned long long* dev_operations;
    int size = sizeof(operations);
    hipMalloc((void**) &dev_operations, size);
    hipMemcpy(dev_operations, &operations, size, hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    float milliseconds;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    sort<<< 1, BLOCK_SIZE >>>(array, dev_operations);
    hipMemcpy(arr, array, ARR_SIZE * sizeof(long int), hipMemcpyDeviceToHost);
    hipMemcpy(&operations, dev_operations, size, hipMemcpyDeviceToHost);

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);


    #if defined(FULL)
        printf("\nОтсортированный массив:\n");
        printf("Массив/Первые 100 его элементов:\n");
        for (long long i = 0; i < sizeForPrintf; ++i) {
            printf("%ld ", arr[i]);
        }
        printf("\n");
        printf("Потребовалось операций: %lld\n", operations);
    #endif

    double seconds = (double) milliseconds / 1000;
    printf("Заняло %f секунд\n", seconds);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
