#define DEBUG

#include "hip/hip_runtime.h"

#include <stdio.h>

#define ARR_SIZE 1000000
#define BLOCK_SIZE 1024

__global__ void sort (long long* data)
{
    long long operationOnThread = ARR_SIZE / BLOCK_SIZE / 2 + 1;

    for (long long i = 0; i < ARR_SIZE / 2 + 1; ++i) {

        for (long long iOperation = 0; iOperation < operationOnThread; ++iOperation) {
            long long realIndex = (BLOCK_SIZE * iOperation + threadIdx.x) * 2;
            long long nextIndex = realIndex + 1;
            if (nextIndex < ARR_SIZE) {
                if (data[realIndex] > data[nextIndex]) {
                    long long tmp = data[realIndex];
                    data[realIndex] = data[nextIndex];
                    data[nextIndex] = tmp;
                }
            }
        }

        __syncthreads();

        for (long long iOperation = 0; iOperation < operationOnThread; ++iOperation) {
            long long realIndex = (BLOCK_SIZE * iOperation + threadIdx.x) * 2 + 1;
            long long nextIndex = realIndex + 1;
            if (nextIndex < ARR_SIZE) {
                if (data[realIndex] > data[nextIndex]) {
                    long long tmp = data[realIndex];
                    data[realIndex] = data[nextIndex];
                    data[nextIndex] = tmp;
                }
            }
        }

        __syncthreads();

    }
}

int main()
{
    long long arr[1000000];

    FILE* f = fopen("../input.txt", "rt");
    long long readIndex = 0;
    long long temp = 0;
    while (fscanf(f, "%lld", &temp) == 1) {
        arr[readIndex] = temp;
        ++readIndex;
    }

    #if defined(DEBUG)
        printf("\nИзначальный массив:\n");
        printf("Размер массива: %lld\n", ARR_SIZE);
        long long sizeForPrintf = 100;
        if (sizeForPrintf > ARR_SIZE) {
            sizeForPrintf = ARR_SIZE;
        }
        printf("Массив/Первые 100 его элементов:\n");
        for (long long i = 0; i < sizeForPrintf; ++i) {
            printf("%lld ", arr[i]);
        }
        printf("\n");
    #endif

    long long* array;
    hipMalloc(&array, ARR_SIZE * sizeof(long long));

    hipMemcpy(array, arr, ARR_SIZE * sizeof(long long), hipMemcpyHostToDevice);


    hipEvent_t start, stop;
    float milliseconds;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);


    sort<<< 1, BLOCK_SIZE >>>(array);
    hipMemcpy(arr, array, ARR_SIZE * sizeof(long long), hipMemcpyDeviceToHost);

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    double seconds = (double) milliseconds / 1000;
    printf("Заняло %f секунд\n", seconds);
    hipEventDestroy(start);
    hipEventDestroy(stop);


    #if defined(DEBUG)
        printf("\nОтсортированный массив:\n");
        printf("Массив/Первые 100 его элементов:\n");
        for (long long i = 0; i < sizeForPrintf; ++i) {
            printf("%lld ", arr[i]);
        }
        printf("\n");
        printf("Потребовалось операций: Потом будет\n");
    #endif

    return 0;
}
