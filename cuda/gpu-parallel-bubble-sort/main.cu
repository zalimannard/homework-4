//#define FULL

#include "hip/hip_runtime.h"

#include <stdio.h>

#define ARR_SIZE 1000
#define BLOCK_SIZE 1024

__global__ void sort (long int* data, unsigned long long* operations)
{
    long int operationOnThread = ARR_SIZE / BLOCK_SIZE / 2 + 1;

    for (long int i = 0; i < ARR_SIZE / 2 + 1; ++i) {
        #if defined(FULL)
            atomicAdd(operations, (unsigned long long) 2);
        #endif
        for (long int iOperation = 0; iOperation < operationOnThread; ++iOperation) {
            #if defined(FULL)
                atomicAdd(operations, (unsigned long long) 2);
            #endif
            long int realIndex = (BLOCK_SIZE * iOperation + threadIdx.x) * 2;
            long int nextIndex = realIndex + 1;
            #if defined(FULL)
                atomicAdd(operations, (unsigned long long) 5);
            #endif
            if (nextIndex < ARR_SIZE) {
                if (data[realIndex] > data[nextIndex]) {
                    long int tmp = data[realIndex];
                    data[realIndex] = data[nextIndex];
                    data[nextIndex] = tmp;
                    #if defined(FULL)
                        atomicAdd(operations, (unsigned long long) 1);
                    #endif
                }
                #if defined(FULL)
                    atomicAdd(operations, (unsigned long long) 1);
                #endif
            }
            #if defined(FULL)
                atomicAdd(operations, (unsigned long long) 1);
            #endif
        }

        __syncthreads();

        for (long int iOperation = 0; iOperation < operationOnThread; ++iOperation) {
            #if defined(FULL)
                atomicAdd(operations, (unsigned long long) 2);
            #endif
            long int realIndex = (BLOCK_SIZE * iOperation + threadIdx.x) * 2 + 1;
            long int nextIndex = realIndex + 1;
            #if defined(FULL)
                atomicAdd(operations, (unsigned long long) 2);
            #endif
            if (nextIndex < ARR_SIZE) {
                if (data[realIndex] > data[nextIndex]) {
                    long int tmp = data[realIndex];
                    data[realIndex] = data[nextIndex];
                    data[nextIndex] = tmp;
                    #if defined(FULL)
                        atomicAdd(operations, (unsigned long long) 3);
                    #endif
                }
                #if defined(FULL)
                    atomicAdd(operations, (unsigned long long) 1);
                #endif
            }
            #if defined(FULL)
                atomicAdd(operations, (unsigned long long) 1);
            #endif
        }

        __syncthreads();

    }
}

int main()
{
    printf("Параллельная сортировка пузырьком на GPU\n");
    printf("Размер массива: %ld\n", ARR_SIZE);

    #if defined(FULL)
        printf("Запуск в медленном режиме\n\n");
    #else
        printf("Запуск в быстром режиме\n\n");
    #endif

    long int arr[ARR_SIZE];

    FILE* f = fopen("../input.txt", "rt");
    long int readIndex = 0;
    long int temp = 0;
    while (fscanf(f, "%ld", &temp) == 1) {
        arr[readIndex] = temp;
        ++readIndex;
    }

    #if defined(FULL)
        printf("\nИзначальный массив:\n");
        long int sizeForPrintf = 100;
        if (sizeForPrintf > ARR_SIZE) {
            sizeForPrintf = ARR_SIZE;
        }
        printf("Массив/Первые 100 его элементов:\n");
        for (long int i = 0; i < sizeForPrintf; ++i) {
            printf("%ld ", arr[i]);
        }
        printf("\n");
    #endif

    long int* array;
    hipMalloc(&array, ARR_SIZE * sizeof(long int));
    hipMemcpy(array, arr, ARR_SIZE * sizeof(long int), hipMemcpyHostToDevice);

    unsigned long long operations = 0;
    unsigned long long* dev_operations;
    int size = sizeof(operations);
    hipMalloc((void**) &dev_operations, size);
    hipMemcpy(dev_operations, &operations, size, hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    float milliseconds;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    sort<<< 1, BLOCK_SIZE >>>(array, dev_operations);
    hipMemcpy(arr, array, ARR_SIZE * sizeof(long int), hipMemcpyDeviceToHost);
    hipMemcpy(&operations, dev_operations, size, hipMemcpyDeviceToHost);

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    #if defined(FULL)
        printf("\nОтсортированный массив:\n");
        printf("Массив/Первые 100 его элементов:\n");
        for (long int i = 0; i < sizeForPrintf; ++i) {
            printf("%ld ", arr[i]);
        }
        printf("\n");
        printf("Потребовалось операций: %lld\n", operations);
    #endif

    double seconds = (double) milliseconds / 1000;
    printf("Заняло %f секунд\n", seconds);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
