#define DEBUG

#include "hip/hip_runtime.h"

#include <stdio.h>

#define ARR_SIZE 5
#define BLOCK_SIZE 1024

__global__ void sort (long long* data)
{
    long long iThread = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    __syncthreads();
    for (long long i = 0; i < ARR_SIZE / 2 + 1; ++i) {
        long long realIndex = iThread * 2;
        long long nextIndex = realIndex + 1;
        if (nextIndex < ARR_SIZE) {
            if (data[realIndex] > data[nextIndex]) {
                long long tmp = data[realIndex];
                data[realIndex] = data[nextIndex];
                data[nextIndex] = tmp;
            }
        }
        __syncthreads();

        ++realIndex;
        ++nextIndex;
        if (nextIndex < ARR_SIZE) {
            if (data[realIndex] > data[nextIndex]) {
                long long tmp = data[realIndex];
                data[realIndex] = data[nextIndex];
                data[nextIndex] = tmp;
            }
        }
        __syncthreads();
    }
}

int main()
{
    long long arr[] = { 9, 8, 4, 9, 3  };
    #if defined(DEBUG)
        printf("\nИзначальный массив:\n");
        printf("Размер массива: %lld\n", ARR_SIZE);
        long long sizeForPrintf = 100;
        if (sizeForPrintf > ARR_SIZE) {
            sizeForPrintf = ARR_SIZE;
        }
        printf("Массив/Первые 100 его элементов:\n");
        for (long long i = 0; i < sizeForPrintf; ++i) {
            printf("%lld ", arr[i]);
        }
        printf("\n");
    #endif

    long long* array;
    hipMalloc(&array, ARR_SIZE * sizeof(long long));

    hipMemcpy(array, arr, ARR_SIZE * sizeof(long long), hipMemcpyHostToDevice);


    hipEvent_t start, stop;
    float milliseconds;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);


    sort<<< ARR_SIZE / BLOCK_SIZE / 2 + 1, BLOCK_SIZE >>>(array);
    hipMemcpy(arr, array, ARR_SIZE * sizeof(long long), hipMemcpyDeviceToHost);

    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    double seconds = (double) milliseconds / 1000;
    printf("Заняло %f секунд\n", seconds);
    hipEventDestroy(start);
    hipEventDestroy(stop);


    #if defined(DEBUG)
        printf("\nОтсортированный массив:\n");
        printf("Массив/Первые 100 его элементов:\n");
        for (long long i = 0; i < sizeForPrintf; ++i) {
            printf("%lld ", arr[i]);
        }
        printf("\n");
        printf("Потребовалось операций: Потом будет\n");
    #endif

    return 0;
}
